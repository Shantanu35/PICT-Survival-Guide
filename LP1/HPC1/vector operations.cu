#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__
void addn(int *da, int n){
    int idx = threadIdx.x;
    int steps = 1;
    while(n > 1){
        if(idx < n/2){
            int first = idx*2*steps;
            int second = first + steps;
            da[first] += da[second];
        }
        steps *= 2;
        n /= 2;
    }

}

__global__
void minn(int *da, int n){
    int idx = threadIdx.x;
    int steps = 1;
    while(n > 1){
        if(idx < n/2){
            int first = idx*2*steps;
            int second = first + steps;
            if(da[first] > da[second]){
                da[first] = da[second];
            }
        }
        steps *= 2;
        n /= 2;
    }

}

__global__
void maxn(int *da, int n){
    int idx = threadIdx.x;
    int steps = 1;
    while(n > 1){
        if(idx < n/2){
            int first = idx*2*steps;
            int second = first + steps;
            if(da[first] < da[second]){
                da[first] = da[second];
            }
        }
        steps *= 2;
        n /= 2;
    }

}

__global__
void stdn(int *da, int n, int add){
    int idx = threadIdx.x;
    int steps = 1;
    da[idx] = pow((da[idx] - add/n), 2);
    da[idx+8] = pow((da[idx+8] - add/n), 2);
    while(n > 1){
        if(idx < n/2){
            int first = idx*2*steps;
            int second = first + steps;
            da[first] += da[second];
        }
        steps *= 2;
        n /= 2;
    }

}


int main(){
    int n = 16;
    int a[n];
    for(int i = 0; i < n; i++){
        a[i] = i;
    }

    int *da;
    hipMalloc((void **)&da, n*sizeof(int));

    int add;
    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    addn<<<1, n/2>>>(da, n);
    hipMemcpy(&add, da, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d ", add);

    int mini;
    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    minn<<<1, n/2>>>(da, n);
    hipMemcpy(&mini, da, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d ", mini);

    int maxi;
    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    maxn<<<1, n/2>>>(da, n);
    hipMemcpy(&maxi, da, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d ", maxi);

    printf("%d ", add/n);

    int std;
    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    stdn<<<1, n/2>>>(da, n, add);
    hipMemcpy(&std, da, sizeof(int), hipMemcpyDeviceToHost);
    std = std/n;
    std = sqrt(std);
    printf("%d", std);
}
