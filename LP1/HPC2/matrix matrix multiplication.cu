#include "stdio.h"
#include <hip/hip_runtime.h>

__global__
void matmul(int *da, int *db, int *dc, int n){
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int sum = 0;
    for(int i = 0; i < n; i++){
        sum += da[idy*n + i] * db[i*n + idx];
    }
    dc[idy * n +idx] = sum;
}

int main(){
    int n = 3;
    int a[n][n], b[n][n], c[n][n];
    int *da, *db, *dc;

    for(int i = 0;i < n; i++){
        for(int j = 0;j < n; j++){
            a[i][j] = i;
            b[i][j] = j;
            c[i][j] = 0;
        }
    }

    hipMalloc((void **)&da, n*n*sizeof(int));
    hipMalloc((void **)&db, n*n*sizeof(int));
    hipMalloc((void **)&dc, n*n*sizeof(int));

    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(b), hipMemcpyHostToDevice);

    dim3 blockDim(n, n);
    matmul<<<1, blockDim>>>(da, db, dc, n);

    hipMemcpy(&c, dc, sizeof(c), hipMemcpyDeviceToHost);

    for(int i = 0;i < n; i++){
        for(int j = 0;j < n; j++){
            printf(" %d", c[i][j]);
        }
    }

}
