#include <stdio.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void vectmul(int *da, int *db, int *dc, int n){
    int tid = threadIdx.x;
    int sum = 0;
    for(int i = 0; i < n; i++){
        sum += da[tid*n + i] * db[i];
    }

    dc[tid] = sum;
}

int main(){
    int n = 3;
    int a[n][n], b[n], c[n];
    int *da, *db, *dc;

    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            a[i][j] = i;
        }
        b[i] = i;
        c[i] = 0;
    }

    hipMalloc((void **)&da, n*n*sizeof(int));
    hipMalloc((void **)&db, n*sizeof(int));
    hipMalloc((void **)&dc, n*sizeof(int));

    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(b), hipMemcpyHostToDevice);

    vectmul<<<1, n>>>(da, db, dc, n);

    hipMemcpy(&c, dc, sizeof(c), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++){
        printf("%d ", c[i]);
    }
}
