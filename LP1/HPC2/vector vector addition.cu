#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void vectadd(int *da, int *db, int *dc){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    dc[tid] = da[tid] + db[tid];
}

int main(){
    int n = 3;
    int a[n], b[n], c[n];
    int *da, *db, *dc;

    for(int i = 0; i < n; i++){
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    hipMalloc((void **)&da, n*sizeof(int));
    hipMalloc((void **)&db, n*sizeof(int));
    hipMalloc((void **)&dc, n*sizeof(int));

    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(b), hipMemcpyHostToDevice);

    vectadd<<<n, 1>>>(da, db, dc);

    hipMemcpy(&c, dc, sizeof(c), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++){
        printf("%d ", c[i]);
    }
}
